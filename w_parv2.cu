#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define WORDLE_WORD_LIST_MAX_COUNT 13000
#define WORDLE_WORD_LIST_FILE_NAME "valid-wordle-words.txt"

#define WORDLE_ANSWER_LIST_MAX_COUNT 2350
#define WORDLE_ANSWER_LIST_FILE_NAME "wordle-answers-alphabetical.txt"

#define VERBOSE 0
#define PRINT_DEVICE_SETTINGS 0
#define REPETITIONS 1000
#define RANDOM_SEED 100

char *answer;
int valid_word_count;

__host__
void guess(char **guess_to_check, char **result);
__host__
void normalize_memory_col(char **memory, int col_index);
__host__
void normalize_memory_row(char **memory, int row_index);
__host__
void print_memory(char **memory, int **char_count, int **char_count_atleast_flag);
__host__
void init_memory(char **memory);
__host__
int solver_par2(char **valids, char **valids_d);

void guess(char **guess_to_check, char **result){
    char word_correct[6];
    char word_guess[6];
    for (int i = 0; i < 5; i++){
        word_correct[i] = answer[i];
        word_guess[i] = (*guess_to_check)[i];
    }
    word_correct[5] = '\0';
    word_guess[5] = '\0';

    if (VERBOSE){
        printf("-----------------------------------------\n");
        printf("%s\n", word_correct);
        printf("%s\n", word_guess);
    }

    for (int i = 0; i < 5; i++){
        if (word_correct[i] == word_guess[i]){
            (*result)[i] = 'G';
            word_correct[i] = '_';
            word_guess[i] = '_';
        }
        else {
            (*result)[i] = '_';
        }
    }

    int j;
    for (int i = 0; i < 5; i++){ // for word_guess
        for (j = 0; j < 5; j++){ // word_correct
            if (i == j || word_guess[i] == '_' || word_correct[j] == '_')
                continue;

            if (word_guess[i] == word_correct[j]){
                (*result)[i] = 'y';
                word_guess[i] = '_';
                word_correct[j] = '_';
            }
        }   
    }

    (*result)[5] = '\0';
}

__host__
void normalize_memory_col(char **memory, int col_index){
    char key_O = '\0';
    for (int i = 0; i < 5; i++){ // Iterate over each memory column
        if ((*memory)[i * 6 + 1 + col_index] == 'O')
            key_O = (*memory)[i * 6 + 0];
    }
    for (int i = 0; i < 5; i++){ // Other indicators in the same column in other rows must be X, unless their key is the same 
        if ((*memory)[i * 6 + 0] == '.'){
            continue;
        }
        if ((*memory)[i * 6 + 0] == key_O && (*memory)[i * 6 + 1 + col_index] != 'O'){
            (*memory)[i * 6 + 1 + col_index] = 'O';
            normalize_memory_row(memory, i);
        }
        else if ((*memory)[i * 6 + 0] != key_O && (*memory)[i * 6 + 1 + col_index] != 'X'){
            (*memory)[i * 6 + 1 + col_index] = 'X';
            normalize_memory_row(memory, i);
        }
    }
}

__host__
void normalize_memory_row(char **memory, int row_index){
    for (int i = 0; i < 5; i++){ // Iterate over each indicator
        if ((*memory)[row_index * 6 + 1 + i] == 'O')
            normalize_memory_col(memory, i);
    }
}

__host__
void normalize_memory(char **memory){
    for (int i = 0; i < 5; i++)
        normalize_memory_row(memory, i);
}

__host__
void print_memory(char **memory, int **char_count, int **char_count_atleast_flag){
    printf("Memory:\n");
    for (int i = 0; i < 5; i++){
        printf("    %d:  ", i);
        for (int j = 0; j < 6; j++){
            printf("%c", (*memory)[i * 6 + j]);
        }

        if (i == 0){
            printf("    ");
            for (int j = 0; j < 26; j++){
                printf("%c", (char)(j + 97));
            }
        }
        if (i == 1){
            printf("    ");
            for (int j = 0; j < 26; j++){
                if ((*char_count_atleast_flag)[j] == 0 || (*char_count)[j] != 0)
                    printf("%d", (*char_count)[j]);
                else 
                    printf(" ");
            }
        }
        if (i == 2){
            printf("    ");
            for (int j = 0; j < 26; j++){
                if ((*char_count_atleast_flag)[j] == 0)
                    printf("=");
                else 
                    printf(" ");
            }
        }

        printf("\n");
    }
}

__host__
void init_memory(char **memory){
    for (int i = 0; i < 5; i++)
        for (int j = 0; j < 6; j++)
            (*memory)[i * 6 + j] = '.';
}

__global__
void compute_score(char *valids_d, char *memory_d, int *char_count_d, int *char_count_atleast_flag_d, int *scores_d, int valid_word_count){
    int word_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Frequencies:
    // https://www3.nd.edu/~busiforc/handouts/cryptography/letterfrequencies.html
    // Frequencies of letters from most to least: eariotnslcudpmhgbfywkvxzjq
    // Python code to generate frequencies: for let in "abcdefghijklmnopqrstuvwxyz": print(26-"eariotnslcudpmhgbfywkvxzjq".index(let), end=", ")
    int frequencies[] = {25, 10, 17, 15, 26, 9, 11, 12, 23, 2, 6, 18, 13, 20, 22, 14, 1, 24, 19, 21, 16, 5, 7, 4, 8, 3};

    if (word_index < valid_word_count){

        int is_valid = 1;
        for (int j = 0; j < 5; j++){ // iterate through the 5 letters
            int letter_as_number = (int)(valids_d[word_index * 5 + j]) - 97;

            // count the number of occurences of the letter
            int occurances = 0;
            for (int k = 0; k < 5; k++){
                int letter_as_number_2 = (int)(valids_d[word_index * 5 + k]) - 97;
                if (letter_as_number == letter_as_number_2)
                    occurances++;
            }

            if ((char_count_atleast_flag_d)[letter_as_number] == 1 && occurances < (char_count_d)[letter_as_number]){
                is_valid = 0;
                break;
            }
            else if ((char_count_atleast_flag_d)[letter_as_number] == 0 && occurances != (char_count_d)[letter_as_number]){
                is_valid = 0;
                break;
            }
        }

        // go through rows of memory 
        for (int j = 0; j < 5; j++){
            char letter_to_test = (memory_d)[j * 6 + 0];
            if (letter_to_test == '.') // row can't be used
                break;

            // key must exist in word (second filter)
            int key_exists = 0;
            for (int k = 0; k < 5; k++){ 
                char letter_in_the_word_to_test = (valids_d)[word_index * 5 + k];
                char indicator = (memory_d)[j * 6 + 1 + k];

                if (key_exists == 0 && letter_in_the_word_to_test == letter_to_test)
                    key_exists = 1;

                if ( // individual X and O indicator testing (third and fourth filter)
                    (letter_in_the_word_to_test == letter_to_test && indicator == 'X') || 
                    (letter_in_the_word_to_test != letter_to_test && indicator == 'O')){
                    is_valid = 0;
                    break;
                }
            }
            if (key_exists == 0)
                is_valid = 0;
        }

        // Compute score
        int score_cumm = 0;
        int letter_done[26];
        for (int j = 0; j < 26; j++)
            letter_done[j] = 0;
        for (int j = 0; j < 5; j++){
            char letter_to_compute_score = (valids_d)[word_index * 5 + j];
            int letter_number = (int)letter_to_compute_score - 97;
            int letter_frequency_score = frequencies[letter_number];

            if (letter_done[letter_number] == 1)
                letter_frequency_score = 0;
            else
                letter_done[letter_number] = 1;
            score_cumm += letter_frequency_score;
        }

        scores_d[word_index] = score_cumm * is_valid;
    }
}

__host__
int get_next_word(char **memory, char **valids_d, int **char_count, int **char_count_atleast_flag){
    // Allocate memory for arrays on device
    char *memory_d;
    int *char_count_d;
    int *char_count_atleast_flag_d;
    int *scores_d;
    int *scores;

    scores = (int*)malloc(valid_word_count * sizeof(int));
    hipMalloc(&memory_d, 6 * 5 * sizeof(char));
    hipMalloc(&char_count_d, 26 * sizeof(int));
    hipMalloc(&char_count_atleast_flag_d, 26 * sizeof(int));
    hipMalloc(&scores_d, valid_word_count * sizeof(int));

    // Copy to device
    hipMemcpy(memory_d, *memory, 6 * 5 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(char_count_d, *char_count, 26 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(char_count_atleast_flag_d, *char_count_atleast_flag, 26 * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 dimBlock(1024, 1, 1);
    dim3 dimGrid(ceil((float)WORDLE_WORD_LIST_MAX_COUNT/1024), 1, 1);

    compute_score<<< dimGrid, dimBlock >>>
    (*valids_d, memory_d, char_count_d, char_count_atleast_flag_d, scores_d, valid_word_count);

    // Get Result
    hipMemcpy(scores, scores_d, valid_word_count * sizeof(int), hipMemcpyDeviceToHost);

    // Get Best Word
    int best_word_index = -1;
    int best_word_score = 0;
    for (int i = 0; i < valid_word_count; i++){
        // printf("%d: %d\n", i, scores[i]);
        if (scores[i] > best_word_score){
            best_word_index = i;
            best_word_score = scores[i];
        }
    }

    // Free all memory
    hipFree(char_count_d);
    hipFree(char_count_atleast_flag_d);
    hipFree(memory_d);
    hipFree(scores);
    free(scores);

    if (VERBOSE)
        printf("Best score %d with index %d\n", best_word_score, best_word_index);

    return best_word_index;
}

__host__
void device_init(char **valids_d, char **valids){
    hipMalloc(valids_d, valid_word_count * 5 * sizeof(char));
    hipMemcpy(*valids_d, *valids, valid_word_count * 5 * sizeof(char), hipMemcpyHostToDevice);
}

__host__
void  device_free(char **valids_d){
    hipFree(*valids_d);
}

__host__
int solver_par2(char **valids, char **valids_d){
    char *colors = (char*)malloc(6 * sizeof(char));
    char *guess_word = (char*)malloc(6 * sizeof(char));
    int *links = (int*)malloc(5 * sizeof(int)); // used for linking memory rows to guess letters
    int i, j, k;

    int try_count = 0;

    int* char_count = (int*)malloc(26 * sizeof(int));
    int* char_count_atleast_flag = (int*)malloc(26 * sizeof(int));
    for (i = 0; i < 26; i++){
        char_count[i] = 0;
        char_count_atleast_flag[i] = 1;
    }

    char* memory = (char*)malloc(6 * 5 * sizeof(char)); // 5 rows, 6 columns
    init_memory(&memory);

    for (i = 0; i < 10; i++){ // 50 is a killswitch
        try_count++;
        int valid_word_index = 0;

        // Calculate next best guess
        if (i == 0)
            valid_word_index = 10184; // Initial guess will always be SLATE
        else
            valid_word_index = get_next_word(&memory, valids_d, &char_count, &char_count_atleast_flag);

        // Guess the word to get new colors and information
        for (j = 0; j < 6; j++)
            guess_word[j] = (*valids)[valid_word_index * 5 + j];
        guess(&guess_word, &colors);
        if (VERBOSE)
            printf("%s\n", colors);

        // If the colors are all green, exit (solved)
        if (strcmp(colors, "GGGGG") == 0)
            break;

        // Reset all links
        for (j = 0; j < 5; j++)
            links[j] = 0; 

        // Iterate over unempty memory rows (memory rows with Os must be first) 
        // (prioritize existing memory rows before creating new rows)
        for (j = 0; j < 5; j++){
            if (memory[j * 6] == '.')
                continue;

            char memory_row_key = memory[j * 6];

            // Link the next O indicator
            int linked = 0;
            for (k = 0; k < 5; k++){
                if (memory[j * 6 + 1 + k] == 'O' && colors[k] == 'G' && links[k] == 0){
                    links[k] = 1;
                    linked = 1;
                    break;
                }
            }

            if (linked == 0){ // If O indicator does not exist, find a letter to link to a dot (.)
                for (k = 0; k < 5; k++){ // Iterate over each guess word letter
                    if (memory[j * 6 + 1 + k] == 'X')
                        continue;

                    if (memory_row_key == guess_word[k] && links[k] == 0){
                        if (colors[k] == 'G'){
                            memory[j * 6 + 1 + k] = 'O';
                            links[k] = 1;
                        }
                        else if (colors[k] == 'y'){
                            memory[j * 6 + 1 + k] = 'X';
                            links[k] = 1;
                        }
                    }
                }
            }
        }

        // Iterate over guess colors to iterate new memory rows
        for (j = 0; j < 5; j++){ 
            char guess_word_letter = guess_word[j];
            char guess_word_color = colors[j];
            if (links[j] == 0){
                int unused_row_index = -1;
                for (k = 0; k < 5; k++) // Iterate over memory rows
                    if (memory[k * 6] == '.'){
                        unused_row_index = k;
                        break;
                    }

                if (guess_word_color == 'G'){
                    memory[unused_row_index * 6 + 1 + j] = 'O';
                    memory[unused_row_index * 6] = guess_word_letter;
                    char_count[(int)guess_word_letter - 97] += 1;
                }
                else if (guess_word_color == 'y'){
                    memory[unused_row_index * 6 + 1 + j] = 'X';
                    memory[unused_row_index * 6] = guess_word_letter;
                    char_count[(int)guess_word_letter - 97] += 1;
                }
                else {
                    if (char_count_atleast_flag[(int)guess_word_letter - 97] == 1)
                        char_count_atleast_flag[(int)guess_word_letter - 97] = 0;
                }
            }
        }
        normalize_memory(&memory);

        if (VERBOSE)
            print_memory(&memory, &char_count, &char_count_atleast_flag);
    }

    if (VERBOSE){
        printf("-----------------------------------------\n");
        printf("Done! (%d tries)\n", try_count);
    }

    free(colors);
    free(links);
    free(guess_word);
    free(memory);
    free(char_count);

    return try_count;
}

int main(){
    // Printing of device properties
    if (PRINT_DEVICE_SETTINGS){
        int dev_count;
        hipGetDeviceCount( &dev_count);
        hipDeviceProp_t dev_prop;
        for (int i = 0; i < dev_count; i++) {
            hipGetDeviceProperties( &dev_prop, i);
            printf("DEVICE NUMBER %d:\n", i + 1);
            printf("    name : %s\n", dev_prop.name);
            printf("    clockRate : %d\n", dev_prop.clockRate);
            printf("    maxBlocksPerMultiProcessor : %d\n", dev_prop.maxBlocksPerMultiProcessor);
            printf("    maxThreadsPerBlock : %d\n", dev_prop.maxThreadsPerBlock);
            printf("    maxThreadsPerMultiProcessor : %d\n", dev_prop.maxThreadsPerMultiProcessor);
            printf("    sharedMemPerBlock : %zu\n", dev_prop.sharedMemPerBlock);
            printf("    sharedMemPerBlockOptin : %zu\n", dev_prop.sharedMemPerBlockOptin);
            printf("    sharedMemPerMultiprocessor : %zu\n", dev_prop.sharedMemPerMultiprocessor);
            printf("    warpSize : %d\n", dev_prop.warpSize);
        }
    }

    // Loading of valid words text file
    FILE *textfile;
    char *valids;
    valids = (char*)malloc(WORDLE_WORD_LIST_MAX_COUNT * 5 * sizeof(char));
    valid_word_count = 0;
    textfile = fopen(WORDLE_WORD_LIST_FILE_NAME, "r");
    if(textfile == NULL)
        return 1;
    char tempArray[10];
    while(fgets(tempArray, 10, textfile)){
        valids[valid_word_count * 5 + 0] = tempArray[0];
        valids[valid_word_count * 5 + 1] = tempArray[1];
        valids[valid_word_count * 5 + 2] = tempArray[2];
        valids[valid_word_count * 5 + 3] = tempArray[3];
        valids[valid_word_count * 5 + 4] = tempArray[4];
        valid_word_count++;
    }
    valids[valid_word_count * 5] = '\0';
    fclose(textfile);

    // Loading of valid answers text file
    FILE *textfileAnswers;
    char *answers;
    answers = (char*)malloc(WORDLE_ANSWER_LIST_MAX_COUNT * 5 * sizeof(char));
    int answers_word_count = 0;
    textfileAnswers = fopen(WORDLE_ANSWER_LIST_FILE_NAME, "r");
    if(textfileAnswers == NULL)
        return 1;
    while(fgets(tempArray, 10, textfileAnswers)){
        answers[answers_word_count * 5 + 0] = tempArray[0];
        answers[answers_word_count * 5 + 1] = tempArray[1];
        answers[answers_word_count * 5 + 2] = tempArray[2];
        answers[answers_word_count * 5 + 3] = tempArray[3];
        answers[answers_word_count * 5 + 4] = tempArray[4];
        answers_word_count++;
    }
    answers[answers_word_count * 5] = '\0';
    fclose(textfileAnswers);

    double total_elapsed = 0;
    double total_tries = 0;
    srand(RANDOM_SEED);

    char *valids_d;
    device_init(&valids_d, &valids);

    for (int i = 0; i < REPETITIONS; i++){
        // Prepare answer
        int r = rand();
        int answer_index = r % answers_word_count;
        answer = (char*)malloc(6 * sizeof(char));
        for (int j = 0; j < 5; j++)
            answer[j] = answers[answer_index * 5 + j];
        answer[5] = '\0';
        
        if (VERBOSE)
            printf("Word is: %s\n", answer);

        // Start solver
        // Start recording time
        // https://www.techiedelight.com/find-execution-time-c-program/
        clock_t begin = clock();

        int tries = solver_par2(&valids, &valids_d);

        // Stop recording time and get elapsed time
        clock_t end = clock();
        total_elapsed += ((double)(end - begin) / CLOCKS_PER_SEC);
        total_tries += tries;

        if (VERBOSE == 0)
        printf("%d. Word: %s (%d tries)\n", i + 1,  answer, tries);
    }

    device_free(&valids_d);

    printf("Total elapsed time: %f\n", total_elapsed);
    printf("Average time per word: %f\n", total_elapsed / REPETITIONS);
    printf("Average tries per word: %f\n", total_tries / REPETITIONS);

    // Free memory
    free(valids);
    free(answer);

    return 0;
}